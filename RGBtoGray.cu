/*
* Provides function to turn RGB image to gray image.
*/

#include <hip/hip_runtime.h>
#include<stdint.h>

// Bullshit mangling off.
extern "C"{

// Function takes in RGB color image size of of width
// and heights and outputs gray image to imageout.
// DO NOT USE SAME MEMORY AREA AS INPUT AND OUTPUT!
__global__ void rgbToGray(uint8_t * __restrict__ image,uint8_t * __restrict__ imageout,uint32_t width,uint32_t height){

	// libṕng manual says that RGB to grayscale can be done
	// with followíng formula: Y=(6968 * R + 23434 * G + 2366 * B)/32768.
	// PNG also has cHRM chunk where one can get weights but don't have
	// time to deal with that.
	// http://www.libpng.org/pub/png/libpng-manual.txt

	uint32_t xindex=blockIdx.x*blockDim.x+threadIdx.x;
	uint32_t yindex=blockIdx.y*blockDim.y+threadIdx.y;

	if(xindex<width && yindex<height){
		// Index to the pixel.
		uint32_t index=yindex*width+xindex;
		imageout[index]=(6968*image[index*3+0]+23434*image[index*3+1]+2366*image[index*3+2])/32768;
	}

}

}
